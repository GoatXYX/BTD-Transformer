#include "hip/hip_runtime.h"
#include "ein.h"

#define TM 4
#define TN 4

__global__ void ein_forward(
    float* core_value, float* rw_head_q, float* w_head_k, float* w_head_v, float* output,
    int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int i = bx * blockDim.x + tx;
    int j = by * blockDim.y + ty;
    int k = bz * blockDim.z + tz;

    int b = k / DIM_K;
    k %= DIM_K;

    if (i < DIM_I && j < DIM_J && k < DIM_K && b < DIM_B) {
        float sum = 0.0;
        int base_q = i * DIM_B * DIM_H;
        int base_k = j * DIM_B * DIM_H;
        int base_v = k * DIM_B * DIM_H;

        for (int h = 0; h < DIM_H; h++) {
            int idx_q = base_q + b * DIM_H + h;
            int idx_k = base_k + b * DIM_H + h;
            int idx_v = base_v + b * DIM_H + h;

            float q = rw_head_q[idx_q];
            float k_val = w_head_k[idx_k];
            float v = w_head_v[idx_v];
            float core = core_value[h];

            sum += core * q * k_val * v;
        }
        int idx_output = i * (DIM_B * DIM_J * DIM_K) + b * (DIM_J * DIM_K) + j * DIM_K + k;
        output[idx_output] = sum;
    }
}

__global__ void ein_backward(
    const float* grad_output,
    const float* core_value,
    const float* rw_head_q,
    const float* w_head_k,
    const float* w_head_v,
    float* grad_core_value,
    float* grad_rw_head_q,
    float* grad_w_head_k,
    float* grad_w_head_v,
    int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int i = bx * blockDim.x + tx;
    int j = by * blockDim.y + ty;
    int k = bz * blockDim.z + tz;

    int b = k / DIM_K;
    k %= DIM_K;

    if (i < DIM_I && j < DIM_J && k < DIM_K && b < DIM_B) {
        float go = grad_output[i * (DIM_B * DIM_J * DIM_K) + b * (DIM_J * DIM_K) + j * DIM_K + k];

        int base_q = i * DIM_B * DIM_H;
        int base_k = j * DIM_B * DIM_H;
        int base_v = k * DIM_B * DIM_H;

        // 提前计算索引位置
        int idx_q, idx_k, idx_v;
        for (int h = 0; h < DIM_H; h++) {
            idx_q = base_q + b * DIM_H + h;
            idx_k = base_k + b * DIM_H + h;
            idx_v = base_v + b * DIM_H + h;

            float q = rw_head_q[idx_q];
            float k_val = w_head_k[idx_k];
            float v = w_head_v[idx_v];
            float core = core_value[h];

            atomicAdd(&grad_core_value[h], go * q * k_val * v);
            atomicAdd(&grad_rw_head_q[idx_q], go * core * k_val * v);
            atomicAdd(&grad_w_head_k[idx_k], go * core * q * v);
            atomicAdd(&grad_w_head_v[idx_v], go * core * q * k_val);
        }
    }
}


void launch_ein_forward(float* core_value, float* rw_head_q, float* w_head_k, float* w_head_v, float* output,
                        int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {
    dim3 grid((DIM_I + TM - 1) / TM, (DIM_J + TN - 1) / TN, (DIM_K + TN - 1) / TN * DIM_B);
    dim3 block(TM, TN, TN);
    ein_forward<<<grid, block>>>(core_value, rw_head_q, w_head_k, w_head_v, output, DIM_I, DIM_J, DIM_K, DIM_B, DIM_H);
}

void launch_ein_backward(
    float* grad_output,
    float* core_value,
    float* rw_head_q,
    float* w_head_k,
    float* w_head_v,
    float* grad_core_value,
    float* grad_rw_head_q,
    float* grad_w_head_k,
    float* grad_w_head_v,
    int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {
    dim3 grid((DIM_I + TM - 1) / TM, (DIM_J + TN - 1) / TN, (DIM_K + TN - 1) / TN * DIM_B);
    dim3 block(TM, TN, TN);
    ein_backward<<<grid, block>>>(grad_output, core_value, rw_head_q, w_head_k, w_head_v, grad_core_value, grad_rw_head_q, grad_w_head_k, grad_w_head_v, DIM_I, DIM_J, DIM_K, DIM_B, DIM_H);
}
