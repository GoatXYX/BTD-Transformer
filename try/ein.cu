#include "hip/hip_runtime.h"
#include "ein.h"

#define TM 4
#define TN 4

__global__ void ein_forward(
    double* core_value, double* rw_head_q, double* w_head_k, double* w_head_v, double* output,
    int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int i = bx * blockDim.x + tx;
    int j = by * blockDim.y + ty;
    int k = bz * blockDim.z + tz;

    int b = k / DIM_K;
    k %= DIM_K;

    if (i < DIM_I && j < DIM_J && k < DIM_K && b < DIM_B) {
        double sum = 0.0;
        int base_q = i * DIM_B * DIM_H;
        int base_k = j * DIM_B * DIM_H;
        int base_v = k * DIM_B * DIM_H;

        for (int h = 0; h < DIM_H; h++) {
            int idx_q = base_q + b * DIM_H + h;
            int idx_k = base_k + b * DIM_H + h;
            int idx_v = base_v + b * DIM_H + h;

            double q = rw_head_q[idx_q];
            double k_val = w_head_k[idx_k];
            double v = w_head_v[idx_v];
            double core = core_value[h];

            sum += core * q * k_val * v;
        }
        int idx_output = i * (DIM_B * DIM_J * DIM_K) + b * (DIM_J * DIM_K) + j * DIM_K + k;
        output[idx_output] = sum;
    }
}

__global__ void ein_backward(
    const double* grad_output,
    const double* core_value,
    const double* rw_head_q,
    const double* w_head_k,
    const double* w_head_v,
    double* grad_core_value,
    double* grad_rw_head_q,
    double* grad_w_head_k,
    double* grad_w_head_v,
    int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int i = bx * blockDim.x + tx;
    int j = by * blockDim.y + ty;
    int k = bz * blockDim.z + tz;

    int b = k / DIM_K;
    k %= DIM_K;

    if (i < DIM_I && j < DIM_J && k < DIM_K && b < DIM_B) {
        double go = grad_output[i * (DIM_B * DIM_J * DIM_K) + b * (DIM_J * DIM_K) + j * DIM_K + k];

        int base_q = i * DIM_B * DIM_H;
        int base_k = j * DIM_B * DIM_H;
        int base_v = k * DIM_B * DIM_H;

        for (int h = 0; h < DIM_H; h++) {
            int idx_q = base_q + b * DIM_H + h;
            int idx_k = base_k + b * DIM_H + h;
            int idx_v = base_v + b * DIM_H + h;

            double q = rw_head_q[idx_q];
            double k_val = w_head_k[idx_k];
            double v = w_head_v[idx_v];
            double core = core_value[h];

            atomicAdd(&grad_core_value[h], go * q * k_val * v);
            atomicAdd(&grad_rw_head_q[idx_q], go * core * k_val * v);
            atomicAdd(&grad_w_head_k[idx_k], go * core * q * v);
            atomicAdd(&grad_w_head_v[idx_v], go * core * q * k_val);
        }
    }
}


void launch_ein_forward(double* core_value, double* rw_head_q, double* w_head_k, double* w_head_v, double* output,
                        int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {
    dim3 grid((DIM_I + TM - 1) / TM, (DIM_J + TN - 1) / TN, (DIM_K + TN - 1) / TN * DIM_B);
    dim3 block(TM, TN, TN);
    ein_forward<<<grid, block>>>(core_value, rw_head_q, w_head_k, w_head_v, output, DIM_I, DIM_J, DIM_K, DIM_B, DIM_H);
}

void launch_ein_backward(
    double* grad_output,
    double* core_value,
    double* rw_head_q,
    double* w_head_k,
    double* w_head_v,
    double* grad_core_value,
    double* grad_rw_head_q,
    double* grad_w_head_k,
    double* grad_w_head_v,
    int DIM_I, int DIM_J, int DIM_K, int DIM_B, int DIM_H) {
    dim3 grid((DIM_I + TM - 1) / TM, (DIM_J + TN - 1) / TN, (DIM_K + TN - 1) / TN * DIM_B);
    dim3 block(TM, TN, TN);
    ein_backward<<<grid, block>>>(grad_output, core_value, rw_head_q, w_head_k, w_head_v, grad_core_value, grad_rw_head_q, grad_w_head_k, grad_w_head_v, DIM_I, DIM_J, DIM_K, DIM_B, DIM_H);
}
